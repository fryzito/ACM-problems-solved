#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h" 
#include "" 
#include <stdio.h> 
#include <stdlib.h> 
 
// declaracion de funciones 
// GLOBAL: funcion llamada desde el host y ejecutada en el device (kernel) 
__global__ void suma_GPU(int a, int b, int* c) { 
    *c = a + b; 
} 

// HOST: funcion llamada y ejecutada desde el host 
__host__ int suma_CPU(int a, int b) { 
    return (a + b); 
} 
 
int main() { 
    
    // declaraciones 
    int n1 = 1, n2 = 2, c = 0; 
    int* hst_c; 
    int m1 = 10, m2 = 20; 
    int* dev_c; 
    // reserva en el host 
    hst_c = (int*)malloc(sizeof(int)); 
    // reserva en el device 
    hipMalloc((void**)&dev_c, sizeof(int)); 
    // llamada a la funcion suma_CPU 
    c = suma_CPU(n1, n2); 
    // resultados CPU 
    printf("CPU:\n"); 
    printf("%2d + %2d = %2d \n", n1, n2, c); 
    // llamada a la funcion suma_GPU 
    suma_GPU << <1, 1 >> > (m1, m2, dev_c); 
    // recogida de datos desde el device 
    hipMemcpy(hst_c, dev_c, sizeof(int), hipMemcpyDeviceToHost); 
    // resultados GPU 
    printf("GPU:\n"); 
    printf("%2d + %2d = %2d \n", m1, m2, *hst_c); 
    // salida 
    printf("\npulsa INTRO para finalizar..."); 
    fflush(stdin); 
    char tecla = getchar(); 
 
    return 0;
} 