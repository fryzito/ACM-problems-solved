#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void helloworld() {
    printf("Hola Mundo..! Soy el hilo con id bloque: {%d %d}, id de hilo: { %d %d %d }\n",
    blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, threadIdx.z);
}

int main() {
    helloworld<<<1, 1>>>(); // Lanza un hilo
    hipDeviceSynchronize(); // Espera que el GPU termine antes de salir
    return 0;
}