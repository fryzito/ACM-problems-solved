#include "hip/hip_runtime.h" 
 
#include <stdio.h> 
#include <stdlib.h> 

__global__ void helloworld()  
{  
 printf("Hola Mundo..! Soy el hilo con id bloque: {%d %d}, id de hilo{ %d %d %d }\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, 
          threadIdx.z);  
} 

int main() { 
    dim3 threads(1, 2, 4); //--nro de hilos por bloque 
    dim3 grid(2, 1); //--nro de bloques por grid 
    helloworld <<< grid, threads >>> (); //--se define un hilo por bloque 
    return 0; 
}